//
// Created by agsa on 13/11/23.
//

#include <iostream>
#include <math.h>

/*** Sigmoid function ***/
__global__ void sigmoid_kernel(float *input, int rows, int cols) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; // row
    int j = blockIdx.x * blockDim.x + threadIdx.x; // col

    if(i < rows && j < cols) {
        int index = i * cols + j;
        input[index] = 1.0/(1.0 + expf(-input[index]));
    }
}

extern "C"
float * sigmoid_of_matrix(float *input, int rows, int cols) {
    const int blocksize = 16;
    unsigned int mem_size_input = sizeof(float) * rows * cols;

    float *d_input;
    hipMalloc((void **) &d_input, mem_size_input);
    hipMemcpy(d_input, input, mem_size_input, hipMemcpyHostToDevice);

    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid((rows-1)/dimBlock.x + 1, ceil(float(cols)/dimBlock.y));
    sigmoid_kernel<<<dimGrid, dimBlock>>>(d_input, rows, cols);
    
    // allocate host memory for the result
    float *h_output = (float *) malloc(mem_size_input);

    hipMemcpy(h_output, d_input, mem_size_input, hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();  // Wait for the kernel to finish
    // TODO : free memory of gpu
    return h_output;
}

/*** Matrix multiplication ***/
__global__ void matrix_mul_kernel(float *C, float *A, float *B, int wA, int hB) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Accumulate row i of A and column j of B
    int i = by * blockDim.y + ty;
    int j = bx * blockDim.x + tx;

    float accu = 0.0;

    for (int k = 0; k < wA; k++) {
        accu = accu + A[i * wA + k] * B[k * hB + j];
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    C[i * hB + j] = accu;
}

extern "C"
float * matrix_multiplication(float *A, int wA, int hA,  float *B, int wB, int hB) {
    const int blocksize = 256;
    float *C;
    if(hA == wB) {
        int size = wA * hB * sizeof(float);
        hipMalloc((void **) &C, size);
        dim3 dimBlock(blocksize, blocksize);
        dim3 dimGrid((wA-1)/dimBlock.x + 1, ceil(float(hB)/dimBlock.y));
        matrix_mul_kernel<<<dimGrid, dimBlock>>>(A, B, C, wA, hB);
        hipDeviceSynchronize();  // Wait for the kernel to finish
    }
    return C;
}

/*** Forward layer ***/
__global__ void forward_layer_kernel(float *C, float *A, float *B, int wA, int hB, float *b) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Accumulate row i of A and column j of B
    int i = by * blockDim.y + ty;
    int j = bx * blockDim.x + tx;

    float accu = 0.0;

    for (int k = 0; k < wA; k++) {
        accu = accu + A[i * wA + k] * B[k * hB + j] + b[j];
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    C[i * hB + j] = accu;
}

extern "C"
float * forward_layer(float *A, int wA, int hA,  float *B, int wB, int hB, float *b, int hb) {
    const int blocksize = 256;
    float *C;
    if(hA == wB && hB == hb) {
        int size = wA * hB * sizeof(float);
        hipMalloc((void **) &C, size);
        dim3 dimBlock(blocksize, blocksize);
        dim3 dimGrid((wA-1)/dimBlock.x + 1, ceil(float(hB)/dimBlock.y));
        forward_layer_kernel<<<dimGrid, dimBlock>>>(A, B, C, wA, hB, b);
        hipDeviceSynchronize();  // Wait for the kernel to finish
    }
    return C;
}

#include <hip/hip_runtime.h>


__global__
void transpose_kernel(float* input, float* output, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols) {
        output[j * rows + i] = input[i * cols + j];
    }
}

extern "C"
void transpose_matrix(float* input, float* output, int rows, int cols) {
    const int block_size = 256;  // Adjust this based on your matrix size
    dim3 dimBlock(block_size, block_size);
    dim3 dimGrid((rows - 1) / dimBlock.x + 1, (cols - 1) / dimBlock.y + 1);

    transpose_kernel<<<dimGrid, dimBlock>>>(input, output, rows, cols);

    hipDeviceSynchronize();  // Wait for the kernel to finish
}
