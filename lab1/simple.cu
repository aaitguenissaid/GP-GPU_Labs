// Assigns every element in an array with its index.
// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple


#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

__global__ void simple(float *c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    c[idx] = pow(threadIdx.x, 0.5);
}

int main() {
    // Define problem size
    const int N = 1024;

    // Define number of blocks
    const int blocksize = 16;

    // Create host and device data strutures
    float *c_h = new float[N];
    float *c_d;

    // Give size of array to allocate on GPU
    const int size = N * sizeof(float);

    //	Allocate array on GP GPU
    hipMalloc((void **) &c_d, size);

    // Define workspace topology
    dim3 dimBlock(blocksize, 1);
    dim3 dimGrid(N / dimBlock.x, 1);

    // Execute kernel
    simple<<<dimGrid, dimBlock>>>(c_d);

    // Wait for kernel completion
    hipDeviceSynchronize();

    // Copy result of computation back on host
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
        std::cout << c_h[i] << " ";

    std::cout << std::endl;

    // Free memory
    hipFree(c_d);
    delete[] c_h;

    std::cout << "done" << std::endl;

    return EXIT_SUCCESS;
}
