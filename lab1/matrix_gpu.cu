// Matrix addition, GPU version


#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add_matrix(float *a, float *b, float *c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    c[index] = a[index] + b[index];
}

int main() {
    const int N = 16;
    const int blocksize = 16;

    float *a = new float[N * N];
    float *b = new float[N * N];
    float *c_h = new float[N * N];
    float *a_d;
    float *b_d;
    float *c_d;


    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++) {
            a[i + j * N] = 10 + i;
            b[i + j * N] = float(j) / N;
        }

    const int size = N * N * sizeof(float);
    hipMalloc((void **) &a_d, size);
    hipMalloc((void **) &b_d, size);
    hipMalloc((void **) &c_d, size);

    hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, size, hipMemcpyHostToDevice);

    dim3 dimBlock(blocksize, 1);
    dim3 dimGrid(N / dimBlock.x, 1);

    add_matrix<<<dimGrid, dimBlock>>>(a_d, b_d, c_d);

    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << c_h[i + j * N] << " ";
        }
        std::cout << std::endl;
    }
}
