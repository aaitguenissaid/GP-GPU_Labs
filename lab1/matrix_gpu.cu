#include "hip/hip_runtime.h"
// Matrix addition, GPU version

#include <iostream>

__global__ void add_matrix(float *a, float *b, float *c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    c[index] = a[index] + b[index];
}

int main() {
    const int N = 16;
    const int blocksize = 16;

    float *a = new float[N * N];
    float *b = new float[N * N];
    float *c_h = new float[N * N];
    float *c_d;

    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++) {
            a[i + j * N] = 10 + i;
            b[i + j * N] = float(j) / N;
        }

    const int size = N * N * sizeof(float);
    hipMalloc((void **) &c_d, size);

    dim3 dimBlock(blocksize, 1);
    dim3 dimGrid(N / dimBlock.x, 1);

    add_matrix<<<dimGrid, dimBlock>>>(a, b, c);

    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << c_h[i + j * N] << " ";
        }
        std::cout << std::endl;
    }
}
