#include "hip/hip_runtime.h"
// Matrix addition, GPU version

#include <iostream>

__global__ void add_matrix(float *a, float *b, float *c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    c[index] = a[index] + b[index];
}

int main() {
    const int N = 16;
    const int blocksize = 16;

    float *a = new float[N * N];
    float *b = new float[N * N];
    float *c;

    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++) {
            a[i + j * N] = 10 + i;
            b[i + j * N] = float(j) / N;
        }

    const int size = N * N * sizeof(float);
    hipMalloc((void **) &c_d, size);

    dim3 dimBlock(blocksize, 1);
    dim3 dimGrid(N / dimBlock.x, 1);

    add_matrix<<<dimGrid, dimBlock>>>(a, b, c);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << c[i + j * N] << " ";
        }
        std::cout << std::endl;
    }
}
